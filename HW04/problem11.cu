#include "hip/hip_runtime.h"
#include <time.h>
#include <stdlib.h>
#include <stdio.h>
#include "randoms.c"
#include <hip/hip_runtime.h>

#define PI 3.1415926535
#define MAX(a,b) (a>b)?a:b
#define MIN(a,b) (a<b)?a:b
#define WEIGHT(R,i) sqrt((double)(2/(PI*R*R)))*__expf(-2*(i-R)*(i-R)/(R*R))


__global__ void kernel1(double *inarray,double *outarray, int R, int N)
{
	int treadid = threadIdx.x+threadIdx.y*blockDim.x+(blockIdx.x+blockIdx.y*gridDim.x)*(blockDim.x*blockDim.y);  
	if(treadid<N){
		double *temp=(double*)malloc((2*R+1)*sizeof(double));
		for(int i = -R;i<R+1;i++)
		{	
			temp[i+R]=inarray[treadid+i]*WEIGHT(R,i);
		}
		int offset = R*2+1;
		while(offset!=0){
			if (offset%2==1){
				offset /=2;
				for(int i =0;i<offset;i++){
					temp[i]+=temp[i+offset];
				}
				temp[0]+=temp[2*offset+1];
			}else{
				offset/=2;
				for(int i =0;i<offset;i++){
					temp[i]+=temp[i+offset];
				}

			}
		}
		outarray[treadid]=temp[0];
		free(temp);
	}
}

__global__ void kernel2(double *inarray,double *outarray, int R, int N)
{
	//	__shared__ double memor[threadIdx.x+42];

	int treadid = threadIdx.x+threadIdx.y*blockDim.x+(blockIdx.x+blockIdx.y*gridDim.x)*(blockDim.x*blockDim.y);  
	if(treadid<N){
		double temp=0;
		for(int i = -R;i<R+1;i++)
		{	
			temp+=inarray[treadid+i]*WEIGHT(R,i);
		}
		outarray[treadid]=temp;
	}
}


void read(int argc,char* arcgv[], int *N, int *R, int *seed ,int *s){
	if( argc == 5){
		*N=atoi(arcgv[1]);
		*R=atoi(arcgv[2]);
		*seed=atoi(arcgv[3]);
		*s=atoi(arcgv[4]);
	}else{
		printf("Argument donnot enough!");
	}
}

int find2times(int num)
{
	int n = 1;
	while(n<num){
		n*=2;
	}
	return n;
}

__global__ void getresult(double *arr,double *result, int N)
{

	int treadid = threadIdx.x+threadIdx.y*blockDim.x+(blockIdx.x+blockIdx.y*gridDim.x)*(blockDim.x*blockDim.y);  
	int offset = N/2;
	if(treadid<N){
		arr[treadid]=arr[treadid]*arr[treadid];
	}
	while(offset!=0){
		if(treadid<offset){
			arr[treadid]+=arr[treadid+offset];
		}
		__syncthreads();
		offset/=2;
	}
	*result = sqrt((double)(arr[0]));
}

int main(int argc,char *argcv[]){
	//	clock_t start, end;
	//	double cpu_time_used;
	//	start = clock();
	int N,R,seed,s;
	read(argc,argcv,&N,&R,&seed,&s);
	int gridlen = MIN(32, find2times(N/32/32));
	int gridwid = find2times(N/32/32/gridlen);
	dim3 grid(gridwid,gridlen,1), block(32,32,1);

	double *inArr = (double*)malloc(N*sizeof(double));	
	double *outArr = (double*)malloc(N*sizeof(double));
	double *devInArr,*devOutArr,*result, *devresult;
	result = (double*)malloc(sizeof(double));
	hipMalloc((void**)&devInArr,sizeof(double)*N);
	hipMalloc((void**)&devOutArr,sizeof(double)*N);
	hipMalloc((void**)&devresult,sizeof(double));

	//random
	random_doubles(inArr,-1.0,1.0,N,seed);
	//copy data to device	
	hipMemcpy(devInArr,inArr,sizeof(double)*N,hipMemcpyHostToDevice);
	//calc 
	if(s==0){	
		kernel1 <<<grid,block>>>(devInArr,devOutArr,R,N);
	}else{
		kernel2 <<<grid,block>>>(devInArr,devOutArr,R,N);
	}
	hipError_t error_check = hipGetLastError();
	if( error_check != hipSuccess ){
		printf("%s\n" , hipGetErrorString( error_check ) );
		system("pause") ;
		return 0 ;
	}    



	//get data from device

	getresult<<<grid,block>>>(devOutArr,devresult,N);
	hipMemcpy(result,devresult,sizeof(double),hipMemcpyDeviceToHost);

	printf("%f",*result);


	free(inArr);
	free(outArr);
	free(result);
	hipFree(devresult);
	hipFree(devInArr);
	hipFree(devOutArr);
	return 0;

}
