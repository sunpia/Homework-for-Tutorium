#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#define imin(a,b) (a<b?a:b)

const int N = 33 * 1024;
const int threadPerBlock = 256;
const int blockPerGrid = imin( 32, (N+threadPerBlock-1) / threadPerBlock );

__global__ void dot( float *a, float *b, float *c)
{
    //共享内存, 每个block都有一份拷贝
printf("run");    
__shared__ float cache[threadPerBlock];
    // thread的索引
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // 共享内存的索引,每个block都有cache, 故只用threadIdx.x即可
    int cacheIdx = threadIdx.x;

    float temp = 0;
    while(tid<N)
    {
        //当前tid的thread负责把tid,和tid间隔threadIdx总量整数倍的向量做乘-加操作.
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }
    // 完成求和之后,当前thread把和放在对应的cache中
    cache[cacheIdx] = temp;
    // 在当前block内做同步操作, 等所有thread都完成乘-加运算之后才能做reduction.
    __syncthreads();

    //reduction, 向量缩减.
    //缩减后的结果在cache[0]里.
    int i = blockDim.x/2;
    while (i!=0)
    {
        if (cacheIdx<i)
        {
            cache[cacheIdx] += cache[cacheIdx + i];

        }
        //同步, 等所有thread都完成了当次缩减了才能做下一次的缩减.
        //书上说: 同步不能放在if里面, 否则报错.
        //经过试验没有报错, 结果正确.
        __syncthreads();
        i /= 2;
    }
    // 一个block输出一个值,即cache[0]. 所以c的长度和block数量相同.
    // 限制cacheIdx == 0是为了只做一次赋值操作,节省时间.
    if (cacheIdx == 0)
    {
        c[blockIdx.x] = cache[0];
    }
    // 没有做剩下的累加操作是因为在CPU上做小批量的累加更加有效.
}

int main(void)
{
    float *a, *b, c, *partial_c;
    float *dev_a, *dev_b, *dev_partial_c;

    //分配CPU端的内存
    a = (float *)malloc( N*sizeof(float) );
    b = (float *)malloc( N*sizeof(float) );
    partial_c = (float *)malloc( blockPerGrid*sizeof(float));

    //分配GPU端的内存
   hipMalloc( (void**)&dev_a, N*sizeof(float));
  hipMalloc( (void**)&dev_b, N*sizeof(float));
  hipMalloc( (void**)&dev_partial_c, blockPerGrid*sizeof(float));

    //将主机内存填入数据
    for (int i=0; i<N; i++)
    {
        a[i] = i;
        b[i] = i*2;
    }

    //将向量a和b拷入GPU
    hipMemcpy( dev_a, a, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy( dev_b, b, N*sizeof(float), hipMemcpyHostToDevice);

    //GPU上做点积运算
    dot<<<blockPerGrid, threadPerBlock>>>(dev_a, dev_b, dev_partial_c);

    //将向量拷入主机
    hipMemcpy( partial_c, dev_partial_c, blockPerGrid*sizeof(float), hipMemcpyDeviceToHost);

    //剩余CPU运算, 求累加和
    c = 0;
    for (int i=0; i<blockPerGrid; i++)
    {
        c += partial_c[i];
    }

    //验证结果是否正确
#define sum_square(x) (x*(x+1)*(2*x+1)/6)
    printf( "Does GPU value %.6g = %.6g?\n",c,
            2 * sum_square( (float)(N-1) ) );
    //释放内存
    hipFree( dev_a );
    hipFree( dev_b );
    hipFree( dev_partial_c);

    free( a );
    free( b );
    free( partial_c);

    return 0;
}
