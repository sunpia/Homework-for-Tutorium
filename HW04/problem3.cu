#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <sys/time.h>
#include <thrust/transform_reduce.h>
#include <stdlib.h>
#include <iostream>
#include <math.h>
#include <thrust/scan.h>
#define FUN(x) exp(sin(x))*cos(x/40)
#include <thrust/iterator/zip_iterator.h>
void read(int argc,char* arcgv[], int *npartitions){
	if( argc == 2){
		*npartitions=atoi(arcgv[1]);
	}else{
		printf("Argument donnot enough!");
	}
}

__host__  __device__ int getCoefficient(int num,int npartitions){
	int coeff[]={17,59,43,49,48};
	if(num<4)
	{
		return coeff[num];
	}else if(num>npartitions-4){
		return coeff[npartitions-num];
	}else
	{return 48;}
}


struct simp
{
	simp(double t){h =(double) (100/t);}
	double h;
	__host__ __device__
	double operator()(thrust::tuple<int,int> t){
		int coe,index;
		thrust::tie(index,coe)=t;
		return coe*FUN((double)(index*h));	
	}
};

int main(int argc,char *argcv[]){
	struct timeval start1,end1;

	int npartitions;
	double cpu_time;
	read(argc,argcv,&npartitions);

	simp calc(npartitions);
  	thrust::plus<double> binary_op;
        double init = 0;




	gettimeofday(&start1,NULL);
	thrust::device_vector<int> coe(npartitions,48);
	coe[0]=17;coe[npartitions-1]=17;
	coe[1]=59;coe[npartitions-2]=59;
	coe[2]=43;coe[npartitions-3]=43;
	coe[4]=49;coe[npartitions-4]=49;

	thrust::device_vector<int> index(npartitions,1);
	index[0]=0;
	thrust::inclusive_scan(index.begin(),index.end(),index.begin());

	double rest=thrust::transform_reduce(
			thrust::make_zip_iterator(thrust::make_tuple(index.begin(), coe.begin())),
			thrust::make_zip_iterator(thrust::make_tuple(index.end(), coe.end())),
			simp(npartitions),init,binary_op);

	rest = rest*100/npartitions/48;
	gettimeofday(&end1,NULL);
	cpu_time = ((double)(end1.tv_usec-start1.tv_usec))/1000000;
	std::cout.precision(15);
	std::cout<<rest-32.121040666358<<'\t'<<rest<<std::endl;
	std::cout.precision(6);
	std::cout<<cpu_time<<std::endl;

//	printf("%.15f %.15f\n",fabs(res-32.121040666358),res);
//	printf("%f ms\n" ,cpu_time);
}

