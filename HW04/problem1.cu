#include "hip/hip_runtime.h"
#include <time.h>
#include <stdlib.h>
#include <stdio.h>
#include "../randoms/randoms.c"
#include <hip/hip_runtime.h>
#define PI 3.1415926535
#define MAX(a,b) (a>b)?a:b
#define MIN(a,b) (a<b)?a:b
#define WEIGHT(R,i) sqrt((double)(2/(PI*R*R)))*__expf(-2*(i-R)*(i-R)/(R*R))


__global__ void kernel1(double *inarray,double *outarray, int R, int N)
{
	int treadid = threadIdx.x+threadIdx.y*blockDim.x+(blockIdx.x+blockIdx.y*gridDim.x)*(blockDim.x*blockDim.y);  
	if(treadid<N){
		double temp=0;
		for(int i = -R;i<R+1;i++)
		{	
			if(treadid+i>=0 && treadid+i<N)
			{	
				temp+=inarray[treadid+i]*WEIGHT(R,i);
			}
		}
		outarray[treadid]=temp;
	}
}

__global__ void kernel2(double *inarray,double *outarray, int R, int N)
{
	const int size = 1024;
	__shared__ double mem[size+42];
	int treadinbox =threadIdx.x+threadIdx.y*blockDim.x;
	int treadid = threadIdx.x+threadIdx.y*blockDim.x+(blockIdx.x+blockIdx.y*gridDim.x)*(blockDim.x*blockDim.y);  
	if(treadid<N){
		mem[21+treadinbox] = inarray[treadid];
		if(treadinbox<R){
			if(treadid-R>0){
				mem[21-R+treadinbox] = inarray[treadid-R];	
			}else{
				mem[21-R+treadinbox] = 0;
			}
		}
		if(treadinbox>size-R-1){
			if(treadid+R<N){
				mem[treadinbox+R+21] = inarray[treadid+R];
			}else{
				mem[R+treadinbox+21] = 0;
			}
		}
	}
	__syncthreads();
	if(treadid<N){
		double temp=0;
		for(int i = -R;i<R+1;i++)
		{	
			temp+=mem[treadinbox+21+i]*WEIGHT(R,i);
		}
		outarray[treadid]=temp;
	}

}


void read(int argc,char* arcgv[], int *N, int *R, int *seed ,int *s){
	if( argc == 5){
		*N=atoi(arcgv[1]);
		*R=atoi(arcgv[2]);
		*seed=atoi(arcgv[3]);
		*s=atoi(arcgv[4]);
	}else{
		printf("Argument donnot enough!");
	}
}

int find2times(int num)
{
	int n = 1;
	while(n<num){
		n*=2;
	}
	return n;
}


int main(int argc,char *argcv[]){
	hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);
	float elapsedTime = 0.0;
	int N,R,seed,s;
	read(argc,argcv,&N,&R,&seed,&s);
	int blockwid = 32;
	int blocklen = 32;
	int gridlen = MIN(512, find2times(N/blockwid/blocklen));
	int gridwid = find2times(N/blockwid/blocklen/gridlen);
	dim3 grid(gridwid,gridlen,1), block(blockwid,blocklen,1);

	double *inArr = (double*)malloc(N*sizeof(double));	
	double *outArr = (double*)malloc(N*sizeof(double));
	double *devInArr,*devOutArr;//*result, *devresult;
	hipMalloc((void**)&devInArr,sizeof(double)*N);
	hipMalloc((void**)&devOutArr,sizeof(double)*N);

	//random
	random_doubles(inArr,-1.0,1.0,N,seed);
	//copy data to device	
	hipMemcpy(devInArr,inArr,sizeof(double)*N,hipMemcpyHostToDevice);
	//calc 
	
	hipEventRecord(start,0);
	if(s==0){	
		kernel1 <<<grid,block>>>(devInArr,devOutArr,R,N);
	}else{
		kernel2 <<<grid,block>>>(devInArr,devOutArr,R,N);
	}
	hipEventRecord(end,0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&elapsedTime,start,end);

    
	hipMemcpy(outArr,devOutArr,N*sizeof(double),hipMemcpyDeviceToHost);
	double res=0;
	for(int i=0;i<N;i++){
		res+=outArr[i]*outArr[i];
	}
	res = sqrt(res);
	//get data from device
	printf("%f\n",res);
	printf( "%f ms\n", elapsedTime ); 

	hipEventDestroy(start);
	hipEventDestroy(end);
	free(inArr);
	free(outArr);
	hipFree(devInArr);
	hipFree(devOutArr);
	return 0;

}
