#include "hip/hip_runtime.h"
#include <time.h>
#include <stdlib.h>
#include <stdio.h>
#include "../randoms/randoms.c"
#include <hip/hip_runtime.h>

#define PI 3.1415926535
#define MAX(a,b) (a>b)?a:b
#define MIN(a,b) (a<b)?a:b
#define WEIGHT(R,i) sqrt((float)(2/(PI*R*R)))*__expf(-2*(i-R)*(i-R)/(R*R))


__global__ void kernel1(float *inarray,float *outarray, int R, int N)
{
	int treadid = threadIdx.x+threadIdx.y*blockDim.x+(blockIdx.x+blockIdx.y*gridDim.x)*(blockDim.x*blockDim.y);  
	if(treadid<N){
		float *temp=(float*)malloc((2*R+1)*sizeof(float));
		for(int i = -R;i<R+1;i++)
		{	
			if(treadid+i>=0)
			{temp[i+R]=inarray[treadid+i]*WEIGHT(R,i);}
			else{temp[i+R] = 0;}
		}
		int offset = R*2+1;
		while(offset!=0){
			if (offset%2==1){
				offset /=2;
				for(int i =0;i<offset;i++){
					temp[i]+=temp[i+offset];
				}
				temp[0]+=temp[2*offset+1];
			}else{
				offset/=2;
				for(int i =0;i<offset;i++){
					temp[i]+=temp[i+offset];
				}

			}
		}
		outarray[treadid]=temp[0];
		free(temp);
	}
}

__global__ void kernel2(float *inarray,float *outarray, int R, int N)
{
	const int size = 1024;
	__shared__ float mem[size+42];
	int treadinbox =threadIdx.x+threadIdx.y*blockDim.x;
	int treadid = threadIdx.x+threadIdx.y*blockDim.x+(blockIdx.x+blockIdx.y*gridDim.x)*(blockDim.x*blockDim.y);  
	if(treadid<N){
		mem[21+treadinbox] = inarray[treadid];
		if(treadinbox<R){
			if(treadid-R>0){
				
				mem[21-R+treadinbox] = inarray[treadid-R];	
			}else{
				
				mem[21-R+treadinbox] = 0;}
		}
		if(treadinbox>N-R){
			if(treadid+R<N){
				mem[treadinbox+R+21] = inarray[treadid+R];	
			}else{mem[R+treadinbox+21] = 0;}
		}
	}
	__syncthreads();
//	if(treadid == 0){
//		for (int i=0;i<size+2*R;i++){	
//			printf("%f\n",mem[i+21-R]);
//		}
//	}	
	if(treadid<N){
		float *temp=(float*)malloc((2*R+1)*sizeof(float));
		for(int i = -R;i<R+1;i++)
		{	
			temp[i+R]=mem[treadinbox+21+i]*WEIGHT(R,i);
		}
		int offset = R*2+1;
		while(offset!=0){
			if (offset%2==1){
				offset /=2;
				for(int i =0;i<offset;i++){
					temp[i]+=temp[i+offset];
				}
				temp[0]+=temp[2*offset+1];
			}else{
				offset/=2;
				for(int i =0;i<offset;i++){
					temp[i]+=temp[i+offset];
				}

			}
		}
		outarray[treadid]=temp[0];
		free(temp);
	}

}


void read(int argc,char* arcgv[], int *N, int *R, int *seed ,int *s){
	if( argc == 5){
		*N=atoi(arcgv[1]);
		*R=atoi(arcgv[2]);
		*seed=atoi(arcgv[3]);
		*s=atoi(arcgv[4]);
	}else{
		printf("Argument donnot enough!");
	}
}

int find2times(int num)
{
	int n = 1;
	while(n<num){
		n*=2;
	}
	return n;
}

__global__ void getresult(float *arr,float *result, int N)
{

	int treadid = threadIdx.x+threadIdx.y*blockDim.x+(blockIdx.x+blockIdx.y*gridDim.x)*(blockDim.x*blockDim.y);  
	int offset = N/2;
	if(treadid<N){
		arr[treadid]=arr[treadid]*arr[treadid];
	}
	while(offset!=0){
		if(treadid<offset){
			arr[treadid]+=arr[treadid+offset];
		}
		__syncthreads();
		offset/=2;
	}
	*result = sqrt((float)(arr[0]));
}

int main(int argc,char *argcv[]){
	//	clock_t start, end;
	//	double cpu_time_used;
	//	start = clock();
	int N,R,seed,s;
	read(argc,argcv,&N,&R,&seed,&s);
	int blockwid = 32;
	int blocklen = 32;
	int gridlen = MIN(64, find2times(N/blockwid/blocklen));
	int gridwid = find2times(N/blockwid/blocklen/gridlen);
	dim3 grid(gridwid,gridlen,1), block(blockwid,blocklen,1);

	float *inArr = (float*)malloc(N*sizeof(float));	
	float *outArr = (float*)malloc(N*sizeof(float));
	float *devInArr,*devOutArr,*result, *devresult;
	result = (float*)malloc(sizeof(float));
	hipMalloc((void**)&devInArr,sizeof(float)*N);
	hipMalloc((void**)&devOutArr,sizeof(float)*N);
	hipMalloc((void**)&devresult,sizeof(float));

	//random
	random_floats(inArr,-1.0,1.0,N,seed);
	//copy data to device	
	hipMemcpy(devInArr,inArr,sizeof(float)*N,hipMemcpyHostToDevice);
	//calc 
	if(s==0){	
		kernel1 <<<grid,block>>>(devInArr,devOutArr,R,N);
	}else{
		kernel2 <<<grid,block>>>(devInArr,devOutArr,R,N);
	}
	hipError_t error_check = hipGetLastError();
	if( error_check != hipSuccess ){
		printf("%s\n" , hipGetErrorString( error_check ) );
		system("pause") ;
		return 0 ;
	}    



	//get data from device

	getresult<<<grid,block>>>(devOutArr,devresult,N);
	hipMemcpy(result,devresult,sizeof(float),hipMemcpyDeviceToHost);

	printf("%f",*result);


	free(inArr);
	free(outArr);
	free(result);
	hipFree(devresult);
	hipFree(devInArr);
	hipFree(devOutArr);
	return 0;

}
