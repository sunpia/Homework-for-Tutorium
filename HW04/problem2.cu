#include "hip/hip_runtime.h"
#include <time.h>
#include <stdlib.h>
#include <stdio.h>
#include "../randoms/randoms.c"
#include <hip/hip_runtime.h>
#define PI 3.1415926535
#define MAX(a,b) (a>b)?a:b
#define MIN(a,b) (a<b)?a:b
#define WEIGHT(R,i) sqrt((double)(2/(PI*R*R)))*__expf(-2*(i-R)*(i-R)/(R*R))
const int blockwid = 32;
const int blocklen = 32;
const int gridlen = 512;
const int gridwid = 32;
dim3 grid(gridwid,gridlen,1), block(blockwid,blocklen,1);

const int N = gridlen*gridwid;
const int arry_size =16000000;
__global__ void kernel1(double *inarray1,double *inarray2, double *outarray)
{
	int treadid = threadIdx.x+threadIdx.y*blockDim.x+(blockIdx.x+blockIdx.y*gridDim.x)*(blockDim.x*blockDim.y); 
	int treadinbox = threadIdx.x+threadIdx.y*blockDim.x;
	__shared__ double mem[1024];	       
	if(treadid<arry_size){	
		mem[treadinbox]=inarray1[treadid]*inarray2[treadid];
	}else{
		mem[treadinbox]=0;
	}
	__syncthreads();
	int offset = 512;
	while(offset!=0){
		if(treadinbox<offset){
			mem[treadinbox]+=mem[treadinbox+offset];
		}
		__syncthreads();
		offset /=2;
	}
	if(treadinbox == 0){
		outarray[blockIdx.x+blockIdx.y*gridDim.x]=mem[0];
	}
}


void read(int argc,char* arcgv[], int *seed, int *target){
	if( argc == 3){
		*seed=atoi(arcgv[1]);
		*target=atoi(arcgv[2]);
	}else{
		printf("Argument donnot enough!");
	}
}


__global__ void getresult(double *arr,double *result)
{	
	const int nummem = gridlen;
	__shared__ double mem[nummem];
	int treadinbox = threadIdx.x+threadIdx.y*blockDim.x;
	int blockid = blockIdx.x;
	int treadid = blockid*(blockDim.x*blockDim.y)+treadinbox;	
	int offset = nummem/2;
	if(treadid == 0){*result = 0;}
	if(treadid<N){
		mem[treadinbox]=arr[treadid];
	}
	__syncthreads();
	while(offset!=0){
		if(treadinbox<offset){
			mem[treadinbox]+=mem[treadinbox+offset];
		}
		__syncthreads();
		offset/=2;
	}
	if(treadinbox == 0){
		//	printf("%d\n",blockid);

		result[blockid] = mem[0];
		//	printf("%f\n",mem[0]);
	}
}

double hostcalc(double *arr1,double *arr2,int num){
	double result=0;
	for(int i =0;i<num;i++){
		result+=arr1[i]*arr2[i];	
	}
	return result;
}



int main(int argc,char *argcv[]){
	int seed,target;
	read(argc,argcv,&seed,&target);

	double *inArr1 = (double*)malloc(arry_size*sizeof(double));	
	double *inArr2 = (double*)malloc(arry_size*sizeof(double));	
	double *outArr = (double*)malloc(arry_size*sizeof(double));
	double *result = (double*)malloc(sizeof(double)*gridwid);

	double *devInArr1,*devInArr2,*devOutArr, *devresult;
	double final_result=0;

	hipMalloc((void**)&devInArr1,sizeof(double)*arry_size);
	hipMalloc((void**)&devInArr2,sizeof(double)*arry_size);
	hipMalloc((void**)&devOutArr,sizeof(double)*N);
	hipMalloc((void**)&devresult,sizeof(double)*gridwid);

	//random
	random_doubles(inArr1,-2.0,2.0,arry_size,seed);
	random_doubles(inArr2,-2.0,2.0,arry_size,seed);

	//copy data to device	
	hipMemcpy(devInArr1,inArr1,sizeof(double)*arry_size,hipMemcpyHostToDevice);
	hipMemcpy(devInArr2,inArr2,sizeof(double)*arry_size,hipMemcpyHostToDevice);

	//calc 
	if(target != 0){
		kernel1 <<<grid,block>>>(devInArr1,devInArr2,devOutArr);
		//get data from device

		getresult<<<gridwid,gridlen>>>(devOutArr,devresult);
		hipMemcpy(result,devresult,sizeof(double)*gridwid,hipMemcpyDeviceToHost);
		for(int i =0; i<gridwid;i++){
			final_result+=result[i];
		}
	}else{
		final_result = hostcalc(inArr1,inArr2,arry_size);
	}
	printf("%f",final_result);

	free(inArr1);
	free(inArr2);
	free(outArr);
	free(result);
	hipFree(devresult);
	hipFree(devInArr1);
	hipFree(devInArr2);
	hipFree(devOutArr);
	return 0;

}
